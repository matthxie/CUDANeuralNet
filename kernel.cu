#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>
#include <stdio.h>
#include <cmath>
#include <vector>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <string>
#include <iostream>
#include <algorithm>


__global__ void linearLayer(float* weights, float* biases,
	float* z_values, float* activation_values,
	int* shape, int shape_length) {

	int id = threadIdx.x;

	int layer_offset_biases = 0;
	int layer_offset_weights = 0;
	int layer_offset_activations = 0;

	for (int shape_index = 0; shape_index < shape_length; shape_index++) {
		if (id < shape[shape_index + 1]) {
			int inputs = shape[shape_index];

			for (int neuron_index = 0; neuron_index < inputs; neuron_index++) {
				z_values[layer_offset_biases + id] += weights[layer_offset_weights + (inputs)*id + neuron_index] *
					activation_values[layer_offset_activations + neuron_index];
			}

			z_values[layer_offset_biases + id] += biases[layer_offset_biases + id];
			activation_values[layer_offset_activations + shape[shape_index] + id] = 1.0 / (1.0 + exp(-z_values[id]));
		}

		layer_offset_biases += shape[shape_index + 1];
		layer_offset_weights += shape[shape_index] * shape[shape_index + 1];
		layer_offset_activations += shape[shape_index];

		__syncthreads();
	}
}

__host__ void normalWeightInitialization(float *&weights, float *&biases, float *&host_z, int n_weights, int n_biases, int n_neurons) {
	hipMalloc((void**)&weights, n_weights * sizeof(float));
	hipMalloc((void**)&biases, n_biases * sizeof(float));
	hipMalloc((void**)&host_z, n_biases * sizeof(float));

	hiprandGenerator_t generator;
	hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(generator, time(0));

	hiprandGenerateNormal(generator, weights, n_weights, 0.0f, 1.0f);
	hiprandGenerateNormal(generator, biases, n_biases, 0.0f, 1.0f);
	hiprandGenerateNormal(generator, host_z, n_neurons, 0.0f, 1.0f);

	hiprandDestroyGenerator(generator);
}

__host__ void xavierWeightInitialization(int* shape, float* weights, float* biases, float* host_z, int n_weights, int n_biases) {

}

void feedForwardNetwork(int *shape, int shape_length, float *output) {
	int n_weights = 0;
	int n_biases = 0;
	int n_neurons = 0;

	for (int shape_index = 0; shape_index < shape_length - 1; shape_index++) {
		n_weights += shape[shape_index] * shape[shape_index + 1];
	}

	for (int shape_index = 1; shape_index < shape_length; shape_index++) {
		n_neurons += shape[shape_index];
	}

	n_biases = n_neurons - shape[0];

	float* host_weights = new float[n_weights] {0.0f};
	float* host_biases = new float[n_biases] {0.0f};
	float* host_activations = new float[n_neurons] {0.0f};
	float* host_z = new float[n_biases] {0.0f};

	//normalWeightInitialization(host_weights, host_biases, host_activations, n_weights, n_biases, n_neurons);

	const size_t bytes_biases = n_biases * sizeof(float);
	const size_t bytes_z = n_biases * sizeof(float);
	const size_t bytes_weights = n_weights * sizeof(float);
	const size_t bytes_activations = n_neurons * sizeof(float);
	const size_t bytes_shape = sizeof(int) * shape_length;

	float* device_weights, * device_biases, * device_z, * device_activations;
	int* device_shape;
	hipMalloc(&device_weights, bytes_weights);
	hipMalloc(&device_biases, bytes_biases);
	hipMalloc(&device_z, bytes_z);
	hipMalloc(&device_activations, bytes_activations);
	hipMalloc(&device_shape, bytes_shape);

	hipMemcpy(device_weights, host_weights, bytes_weights, hipMemcpyHostToDevice);
	hipMemcpy(device_biases, host_biases, bytes_biases, hipMemcpyHostToDevice);
	hipMemcpy(device_z, host_z, bytes_z, hipMemcpyHostToDevice);
	hipMemcpy(device_activations, host_activations, bytes_activations, hipMemcpyHostToDevice);
	hipMemcpy(device_shape, shape, bytes_shape, hipMemcpyHostToDevice);

	int n_threads = *std::max_element(shape, shape + shape_length);
	linearLayer << <1, n_threads >> > (device_weights, device_biases, device_z, device_activations, device_shape, shape_length);

	hipMemcpy(host_activations, device_activations, bytes_activations, hipMemcpyDeviceToHost);
	hipMemcpy(host_z, device_z, bytes_z, hipMemcpyDeviceToHost);

	hipFree(device_weights);
	hipFree(device_biases);
	hipFree(device_z);
	hipFree(device_activations);
	hipFree(device_shape);

	output = host_activations;

	int activations_offset = shape[0]; // Skip input values	
	for (int shape_index = 1; shape_index < shape_length; shape_index++)
	{
		std::cout << "Activations " << shape_index << ". hidden layer" << std::endl;

		for (int neuron_nr = 0; neuron_nr < shape[shape_index]; neuron_nr++)
		{
			std::cout << host_activations[neuron_nr + activations_offset] << std::endl;
		}
		activations_offset += shape[shape_index];
	}

	getchar();
}

void backPropagation() {

}