#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>
#include <stdio.h>
#include <cmath>
#include <vector>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <string>
#include <iostream>
#include <algorithm>


__global__ void linearLayer(float* weights, float* biases,
	float* z_values, float* activation_values,
	int* shape, int shape_length) {

	int id = threadIdx.x;

	int layer_offset_z = 0;
	int layer_offset_biases = 0;
	int layer_offset_weights = 0;
	int layer_offset_activations_input = 0;
	int layer_offset_activations_current = shape[0] * blockDim.y;

	for (int shape_index = 0; shape_index < shape_length; shape_index++) {
		if (id < shape[shape_index + 1]) {
			int n_layer_inputs = shape[shape_index];
			int layer_size = shape[shape_index + 1];

			for (int neuron_index = 0; neuron_index < n_layer_inputs; neuron_index++) {
				z_values[layer_offset_biases + threadIdx.y * layer_size + id] += weights[layer_offset_weights + (n_layer_inputs)*id + neuron_index] *
					activation_values[layer_offset_activations_input + threadIdx.y * n_layer_inputs + neuron_index];
			}

			z_values[layer_offset_biases + threadIdx.y * layer_size + id] += biases[layer_offset_biases + id];
			activation_values[layer_offset_activations_current + threadIdx.y * layer_size + id] = 1.0 / 
				(1.0 + exp(-z_values[layer_offset_z + threadIdx.y * layer_size + id]));
		}

		layer_offset_z += shape[shape_index + 1] * blockDim.y;
		layer_offset_biases += shape[shape_index + 1];
		layer_offset_weights += shape[shape_index] * shape[shape_index + 1];
		layer_offset_activations_input = layer_offset_activations_current;
		layer_offset_activations_current += shape[shape_index + 1] * blockDim.y;

		__syncthreads();
	}
}

void normalWeightInitialization(float *&weights, float *&biases, float *&host_z, int n_weights, int n_biases, int n_neurons) {
	hipMalloc((void**)&weights, n_weights * sizeof(float));
	hipMalloc((void**)&biases, n_biases * sizeof(float));
	hipMalloc((void**)&host_z, n_biases * sizeof(float));

	hiprandGenerator_t generator;
	hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(generator, time(0));

	hiprandGenerateNormal(generator, weights, n_weights, 0.0f, 1.0f);
	hiprandGenerateNormal(generator, biases, n_biases, 0.0f, 1.0f);
	hiprandGenerateNormal(generator, host_z, n_neurons, 0.0f, 1.0f);

	hiprandDestroyGenerator(generator);
}

void xavierWeightInitialization(int* shape, float* weights, float* biases, float* host_z, int n_weights, int n_biases) {

}

void feedForwardNetwork(int *shape, int shape_length, float *output, int batch_size) {
	int n_weights = 0;
	int n_biases = 0;
	int n_neurons = 0;

	for (int shape_index = 0; shape_index < shape_length - 1; shape_index++) {
		n_weights += shape[shape_index] * shape[shape_index + 1];
	}

	for (int shape_index = 1; shape_index < shape_length; shape_index++) {
		n_neurons += shape[shape_index];
	}

	n_biases = n_neurons - shape[0];

	float* host_weights = new float[n_weights] {0.0f};
	float* host_biases = new float[n_biases] {0.0f};
	float* host_activations = new float[n_neurons*batch_size] {0.0f};
	float* host_z = new float[n_biases*batch_size] {0.0f};

	//normalWeightInitialization(host_weights, host_biases, host_activations, n_weights, n_biases, n_neurons);

	const size_t bytes_biases = n_biases * sizeof(float);
	const size_t bytes_z = n_biases * sizeof(float);
	const size_t bytes_weights = n_weights * batch_size * sizeof(float);
	const size_t bytes_activations = n_neurons * batch_size * sizeof(float);
	const size_t bytes_shape = sizeof(int) * shape_length;

	float* device_weights, * device_biases, * device_z, * device_activations;
	int* device_shape;
	hipMalloc(&device_weights, bytes_weights);
	hipMalloc(&device_biases, bytes_biases);
	hipMalloc(&device_z, bytes_z);
	hipMalloc(&device_activations, bytes_activations);
	hipMalloc(&device_shape, bytes_shape);

	hipMemcpy(device_weights, host_weights, bytes_weights, hipMemcpyHostToDevice);
	hipMemcpy(device_biases, host_biases, bytes_biases, hipMemcpyHostToDevice);
	hipMemcpy(device_z, host_z, bytes_z, hipMemcpyHostToDevice);
	hipMemcpy(device_activations, host_activations, bytes_activations, hipMemcpyHostToDevice);
	hipMemcpy(device_shape, shape, bytes_shape, hipMemcpyHostToDevice);

	int n_threads = *std::max_element(shape + 1, shape + shape_length);
	dim3 thread_dimensions(n_threads, batch_size);

	linearLayer << <1, thread_dimensions >> > (device_weights, device_biases, device_z, device_activations, device_shape, shape_length);

	hipMemcpy(host_activations, device_activations, bytes_activations, hipMemcpyDeviceToHost);
	hipMemcpy(host_z, device_z, bytes_z, hipMemcpyDeviceToHost);

	hipFree(device_weights);
	hipFree(device_biases);
	hipFree(device_z);
	hipFree(device_activations);
	hipFree(device_shape);

	output = host_activations;

	int activations_offset = shape[0];
	for (int shape_index = 1; shape_index < shape_length; shape_index++)
	{
		std::cout << "Activations " << shape_index << ". hidden layer" << std::endl;

		for (int neuron_nr = 0; neuron_nr < shape[shape_index]; neuron_nr++)
		{
			std::cout << host_activations[neuron_nr + activations_offset] << std::endl;
		}
		activations_offset += shape[shape_index];
	}

	getchar();
}

void backPropagation() {

}